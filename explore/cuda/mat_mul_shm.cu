
#include <hip/hip_runtime.h>
#include <stdio.h>

// Thread block size
#define BLOCK_SIZE 16

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
    int width;
    int height;
    int stride;
    float* elements;
} Matrix;

void print_matrix(const Matrix mat) {
    for(int r=0; r<mat.height; ++r) {
        for(int c=0; c<mat.width; ++c) {
            printf("%.2f\t", mat.elements[mat.width * r + c]);
        }
        printf("\n");
    }
}

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col, float value)
{
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
    return Asub;
}

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = d_A.stride = A.width;
    d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = d_B.stride = B.width;
    d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
    hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = d_C.stride = C.width;
    d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); // 16x16
    dim3 dimGrid((B.width + dimBlock.x -1) / dimBlock.x, (A.height + dimBlock.y -1) / dimBlock.y); // ceil to 2x2
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // warps divergence, this thread if not in the matrix
    if(blockCol * blockDim.x + col >= A.width) return;

    // Shared memory used to store Asub and Bsub respectively
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    int steps = ((A.width + BLOCK_SIZE -1) / BLOCK_SIZE);
    for (int m = 0; m < steps; ++m) {

        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();
        // Multiply Asub and Bsub together
        int width = (steps - m > 1) ? BLOCK_SIZE : A.width % BLOCK_SIZE;
        for (int e = 0; e < width; ++e) {
            Cvalue += As[row][e] * Bs[e][col];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}


int main(int argc, char*argv[]) {
    Matrix mat_A;
    mat_A.height = 30;
    mat_A.width = 40;
    mat_A.stride = mat_A.width;
    mat_A.elements = (float*)malloc(1200 * sizeof(float));
    for(int i=0; i<1200; ++i)mat_A.elements[i] = 1.0;

    Matrix mat_B;
    mat_B.height = 40;
    mat_B.width = 30;
    mat_B.stride = mat_B.width;
    mat_B.elements = (float*)malloc(1200 * sizeof(float));
    for(int i=0; i<1200; ++i)mat_B.elements[i] = 1.0;

    Matrix mat_C;
    mat_C.width = 30;
    mat_C.height = 30;
    mat_C.stride = mat_C.width;
    mat_C.elements = (float*)malloc(900 * sizeof(float));

    MatMul(mat_A, mat_B, mat_C);

    print_matrix(mat_C);
    return 0;
}
