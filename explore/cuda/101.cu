
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define N 10

__global__ void VecAdd(float* A, float* B, float* C) {
    int i = threadIdx.x;
    printf("tid: x=%d\n", i);
    C[i] = A[i] + B[i];
}


int main() {
    float A[N] = {0.0, 1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0};
    float B[N] = {0.0, 1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0};
    float *C = (float*)malloc(sizeof(float) * N);

    float *dA, *dB, *dC;
    hipMalloc((void**)&dA, N * sizeof(float));
    hipMalloc((void**)&dB, N * sizeof(float));
    hipMalloc((void**)&dC, N * sizeof(float));

    hipMemcpy(dA, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, N * sizeof(float), hipMemcpyHostToDevice);
    // Kernel invocation with N threads
    VecAdd<<<1, N>>>(dA, dB, dC);

    hipMemcpy(C, dC, N * sizeof(float), hipMemcpyDeviceToHost);

    for(int i=0; i<N; i++) {
        printf("%d: %f\n", i, *(C+i));
    }

    free(C);

    return 0;
}
