
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define N 10

__global__ void VecAssign(float* A, float *B) {
    int i = threadIdx.x;
    A[i] = 10.0 * i;
    B[i] = 20.0 * i; // bad: B is not alloced by CUDA
}


int main() {
    float *xA, *xB;
    hipHostMalloc(&xA, N * sizeof(float), hipHostMallocDefault);
    printf("uva ptr=%p\n", xA);

    xB = (float*)malloc(sizeof(float) * N);
    printf("host ptr=%p\n", xB);

    printf("------------- set value on host...\n");
    for(int i=0; i<N; i++) {
        *(xA + i) = i * 1.0;
        *(xB + i) = i * 2.0;
        printf("uva  %d: %f\n", i, *(xA+i));
        printf("host %d: %f\n", i, *(xB+i));
    }

    printf("------------- call kernel...\n");
    VecAssign<<<1, N>>>(xA, xB);
    for(int i=0; i<N; i++) {
        printf("uva  %02d: %f\n", i, *(xA+i));
        printf("host %02d: %f\n", i, *(xB+i));
    }

    hipHostFree(xA);
    free(xB);

    getchar();
    return 0;
}
